#include <hip/hip_runtime.h>
#include <stdint.h>
extern "C" __global__ void vec_add(const float* a, const float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

extern "C" void launch_vec_add(const float* a, const float* b, float* c, int n) {
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    vec_add<<<blocks, threads>>>(a, b, c, n);
    hipDeviceSynchronize();
}
